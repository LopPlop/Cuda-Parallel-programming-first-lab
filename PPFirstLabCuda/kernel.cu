#include "hip/hip_runtime.h"
﻿#include <>
#include <hip/hip_runtime.h>

#include <cstdio>
#include<omp.h>

#define _CRT_SECURE_NO_WARNINGS

// Функция возведения в степень
__device__ long long Pow(long long num, int degree) {

    if (degree < 0)                // Ф-ция не предусматривает возведения в отрицательную степень
        return 1;

    if (degree == 0)
        return 1;

    long long tmp = num;
    for (int i = 0; i < degree - 1; i++)
        tmp *= num;
    return tmp;
}

// Функция нахождения произведения десятичных разрядов числа
__device__ long long GetMultiplicationOfDigit(long long num) {
    long long tmp = num;
    long long _tmp = num;
    int cnt = 0;
    long long multiplication = 1;

    while (tmp > 1) {
        tmp /= 10;
        cnt++;
    }

    while (cnt > -1) {
        tmp = _tmp;
        _tmp = tmp / Pow(10, cnt);

        if (_tmp != 0)
            multiplication *= _tmp;
        _tmp = tmp % Pow(10, cnt);
        cnt--;
    }

    return multiplication;
}

// Занимается поиском степени, если не найдена, возвращается 0
__device__ int FindDegree(float num) {
    int tmp;
    for (int i = 2; i < 9; i++) {
        float a = 1.0 / (float)i;
        tmp = (int)powf(num, a);
        if (num == powf(tmp, (float)i)) return i;
    }
    return 0;
}

__global__ void CudaCalculations(long long N_) {
    long long N = N_ + threadIdx.x + 1;
    int tmp = 0;
        tmp = FindDegree(GetMultiplicationOfDigit(N));
        if (tmp != 0) {
            printf("Decision: degree - %d\nnumber - %lld; ", tmp, N);
            return;
        }
}

__global__ void CudaCalculations1(long long N) {
    long long _num = N + 1;

    while (1) {
        _num++;
        if (FindDegree(GetMultiplicationOfDigit(_num)) != 0) {
            printf("Decision: degree - %d, number - %d; ", FindDegree(GetMultiplicationOfDigit(_num)), _num);
            return;
        }
    }
}


int main(int args, const char* argv[]) {
    hipEvent_t start, stop;
    float gpuTime = 0.0;
    long long N;
    printf("Enter N: ");
    scanf("%lld", &N);

    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start, 0);

    //CudaCalculations <<<1, 1>>> (N);
    CudaCalculations1 << <1, 1 >> > (N);

    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&gpuTime, start, stop);

    printf("Time: %f; ", gpuTime);

    return 0;
}
